
#include "ttm.cuh"

thrust::device_vector<value_t>
contract_first_mode(const CSFTensor3 &tensor, const DenseMatrix &matrix) {
  size_t out_num_chunks = tensor.fidx[1].size();
  size_t out_chunk_size = matrix.ncols;
  thrust::device_vector<value_t> out_values(out_num_chunks * out_chunk_size);
  out_values.resize(out_num_chunks * out_chunk_size);

  const index_t max_nblocks = 32768;
  const index_t max_nthreads_per_block = 256;
  index_t max_nthreadsy = 32;

  index_t nthreadsx = 1;
  index_t nthreadsy = 1;
  index_t all_nblocks = 0;
  index_t nblocks = 0;
  index_t shmen_size = 0;

  if (matrix.ncols <= max_nthreadsy)
    nthreadsx = matrix.ncols;
  else
    nthreadsx = max_nthreadsy;
  nthreadsy = max_nthreads_per_block / nthreadsx;

  if (out_num_chunks < nthreadsy) {
    nthreadsy = out_num_chunks;
    nblocks = 1;
  } else {
    all_nblocks = (out_num_chunks + nthreadsy - 1) / nthreadsy;
    if (all_nblocks < max_nblocks) {
      nblocks = all_nblocks;
    } else {
      nblocks = max_nblocks;
    }
  }
  shmen_size = nthreadsx * nthreadsy * sizeof(value_t);
  assert(shmen_size >= nthreadsx * nthreadsy * sizeof(value_t));
  dim3 dimBlock(nthreadsx, nthreadsy);
  // std::cout << "SHMEM size: " << (shmen_size / sizeof(value_t)) << " ("
  //           << shmen_size << " bytes)\n";
  // std::cout << "all_nblocks: " << all_nblocks << "; nthreadsx: " << nthreadsx
  //           << "; nthreadsy: " << nthreadsy << '\n';

  // std::cout << "U_rows: " << matrix.nrows << ": U_cols: " << matrix.ncols
  //           << '\n';
  // std::cout << "out_num_chunks = " << out_num_chunks
  //           << "; out_chunk_size: " << out_chunk_size << '\n';
  GPUTimer timer;
  timer.start();

  spt_TTMRankRBNnzKernelSM<<<nblocks, dimBlock, shmen_size>>>(
      thrust::raw_pointer_cast(out_values.data()), out_chunk_size, out_num_chunks,
      thrust::raw_pointer_cast(tensor.d_values.data()), thrust::raw_pointer_cast(tensor.fidx[2].data()),
      thrust::raw_pointer_cast(tensor.fptr[1].data()), thrust::raw_pointer_cast(matrix.d_values.data()),
      matrix.nrows, matrix.ncols, matrix.ncols
  );
  auto time = timer.seconds();
  fmt::print(
      "First_contraction<<<{}, ({}, {}), {}>>>:\n\t  executed in {} [s]\n", nblocks,
      dimBlock.x, dimBlock.y, shmen_size, time
  );
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
  return out_values;
}

thrust::device_vector<value_t> contract_second_mode(
    const CSFTensor3 &tensor, const DenseMatrix &matrix,
    const thrust::device_vector<value_t> &in_values, const size_t subchunk_size
) {
  size_t out_num_chunks = tensor.fidx[0].size();
  size_t out_chunk_size = matrix.ncols * subchunk_size;
  // fmt::print("\nnum_chunks = {}; chunk_size = {}; matrix.ncols = {}; subchunk_size = {}\n", out_num_chunks, out_chunk_size, matrix.ncols, subchunk_size);
  thrust::device_vector<value_t> out_values(out_num_chunks * out_chunk_size);
  auto threads = dim3(matrix.ncols, 1024 / matrix.ncols);
  auto grid = dim3(
    out_num_chunks
    // (subchunk_size + threads.x - 1) / threads.x,
    // (out_num_chunks + threads.y - 1) / threads.y
  );

  GPUTimer timer;
  timer.start();
  // fmt::print("U = {}\n", matrix.d_values);
      
  ttm_semisparse_kernel<<<grid, threads>>>(
      thrust::raw_pointer_cast(tensor.fptr[0].data()), thrust::raw_pointer_cast(tensor.fidx[1].data()),
      matrix.nrows, matrix.ncols, out_num_chunks, out_chunk_size, subchunk_size,
      thrust::raw_pointer_cast(out_values.data()), thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(matrix.d_values.data())
  );
  auto time = timer.seconds();
  fmt::print(
      "Second_contraction<<<({}, {}),({}, {})>>>\n\t  executed in {} [s]\n", grid.x,
      grid.y, threads.x, threads.y, time
  );
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
  return out_values;
}

thrust::device_vector<value_t> contract_last_mode(
    const CSFTensor3 &tensor, const std::vector<DenseMatrix> &matrices,
    const thrust::device_vector<value_t> &in_values, const size_t subchunk_size
) {
  auto mode = tensor.cyclic_permutation.front();
  const auto& matrix = matrices[mode];

  size_t out_num_chunks = 1;
  size_t out_chunk_size = matrix.ncols * subchunk_size;
  // fmt::print("\nnum_chunks = {}; chunk_size = {}; matrix.ncols = {}; subchunk_size = {}\n", out_num_chunks, out_chunk_size, matrix.ncols, subchunk_size);
  thrust::device_vector<value_t> out_values(out_num_chunks * out_chunk_size);
  auto threads = dim3(matrix.ncols, 1024 / matrix.ncols);
  auto grid = dim3(
    out_num_chunks
    // (subchunk_size + threads.x - 1) / threads.x,
    // (out_num_chunks + threads.y - 1) / threads.y
  );

  GPUTimer timer;
  timer.start();

  // Only one fiber left
  thrust::host_vector<index_t> h_fptr(2, 0);
  h_fptr.back() = tensor.fidx[0].size();
  // fmt::print("\nh_fptr = {}\n", h_fptr);
  auto d_fptr = thrust::device_vector<index_t>(h_fptr);
  // thrust::sequence(d_fptr.begin(), d_fptr.end());
      
  ttm_semisparse_kernel<<<grid, threads>>>(
      thrust::raw_pointer_cast(d_fptr.data()), thrust::raw_pointer_cast(tensor.fidx[0].data()),
      matrix.nrows, matrix.ncols, out_num_chunks, out_chunk_size, subchunk_size,
      thrust::raw_pointer_cast(out_values.data()), thrust::raw_pointer_cast(in_values.data()),
      thrust::raw_pointer_cast(matrix.d_values.data())
  );
  auto time = timer.seconds();
  fmt::print(
      "Last_contraction<<<({},({}, {})>>>:\n  executed in {} [s]\n", grid.x,
      threads.x, threads.y, time
  );
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    printf("Error: %s\n", hipGetErrorString(err));
  return out_values;
}

thrust::host_vector<value_t> ttm_chain(
    const CSFTensor3 &tensor, std::vector<DenseMatrix> &factor_matrices
) {
  thrust::host_vector<index_t> last_mode(tensor.fidx[0]);

  // Contract First Mode
  auto out_first = contract_first_mode(
      tensor, factor_matrices[tensor.cyclic_permutation.back()]
  );
  // fmt::print("out_first = {}\n", out_first);
  auto subchunk_size = factor_matrices[tensor.cyclic_permutation.back()].ncols;
  // Contract Second Mode
  auto out_second = contract_second_mode(
      tensor,
      factor_matrices[tensor.cyclic_permutation[1]],
      out_first, subchunk_size
  );

  subchunk_size *= factor_matrices[tensor.cyclic_permutation[1]].ncols;
  auto &U_update = factor_matrices[tensor.cyclic_permutation.front()];

  thrust::host_vector<value_t> ssp_matrix(out_second);
  return ssp_matrix;
}