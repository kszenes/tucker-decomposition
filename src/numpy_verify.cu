#include "hip/hip_runtime.h"
// Prints to file a python script that verifies result

#include "numpy_verify.cuh"

void print_verification_script(
  const COOTensor3 &X,
  const std::vector<CSFTensor3>& CSFTensors,
  const std::vector<DenseMatrix>& factor_matrices,
  const thrust::device_vector<value_t>& coreTensor
) {
  const std::string filename{"verify.py"};
  fmt::print("Printing verification script to {}\n", filename);
  std::ofstream output(filename);
  output << fmt::format("import numpy as np\n");
      int i = 0;
      for (const auto& e : X.d_modes) {
        output << fmt::format("\nmode{} = np.array({})\n", i, e);
        ++i;
      }
      output << fmt::format("\nvals = np.array({})\n", X.d_values);
      output << fmt::format("\nX = np.zeros({})\n",
        CSFTensors.front().shape);
      output << fmt::format("\nX[mode0, mode1, mode2] = vals\n");
      output << fmt::format("# Factor matrices:\n");
      i = 0;
      for (const auto& e : factor_matrices) {
        output << fmt::format("\nU{} = np.array({}).reshape({},{})\n",
          i, e.d_values, e.nrows, e.ncols);
        ++i;
      }
      output << fmt::format("\ncore = np.array({}).reshape({}, {}, {})\n",
        coreTensor,
        factor_matrices[0].ncols,
        factor_matrices[1].ncols,
        factor_matrices[2].ncols);

      // output << fmt::format("core = core.transpose([2, 1, 0])\n");
      output << fmt::format("out = np.einsum('ijk,li,mj,nk->lmn', core, U0, U1, U2)\n");
      output << fmt::format("to_X = np.linalg.norm(X - out) / np.linalg.norm(X)\n");
      output << fmt::format("print('to_X: ', to_X)\n");
      output << fmt::format("core2 = np.einsum('ijk,il,jm,kn->lmn', X, U0, U1, U2)\n");
      output << fmt::format("to_core = np.linalg.norm(core - core2) / np.linalg.norm(core)\n");
      output << fmt::format("print('to_core: ', to_core)\n");
}